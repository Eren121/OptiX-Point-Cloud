#include "hip/hip_runtime.h"
#include <optix.h>
#include "ray.cuh"
#include "Record.hpp"
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"
#include "Point.h"

texture<uchar4, 2, hipReadModeElementType> texRef;

extern "C"
{
    __global__ void __raygen__my_program()
    {
        const uint2 idx = make_uint2(optixGetLaunchIndex());
        const uint2 dim = make_uint2(optixGetLaunchDimensions());
        
        const bool perspective = true;

        // Comme en OpenGL, on définit la taille de la fenêtre dans l'intervalle [-1;1]
        // Et on lance un rayon vers l'intérieur de la fenêtre donc en -Z
        float2 gridPos = (make_float2(idx) / make_float2(dim) * 2.0f) - 1.0f;
        float3 rayDirection, rayOrigin;

        if(perspective)
        {
            // Variable utile pour calculer la direction du rayon
            // Ici, on considère que:
            // A la distance znear, le champ de vision voit exactement la taille du viewport
            // définie par (length(params.u), length(params.v))
            const float znear = 1.0f;

            // L'origine du rayon est toujours l'origine de la caméra pour une perspective
            rayOrigin = params.camera.origin;

            
            // Point projecté à une distance znear
            const float3 target =
                gridPos.x * params.camera.u
              + gridPos.y * params.camera.v
              + params.camera.direction * znear;

            rayDirection = normalize(target);
        }
        else
        {
            // Ortographique

            rayDirection = params.camera.direction;

            // Passe de l'intervalle [-1;1] aux coordonnées caméra pour ce pixel
            rayOrigin = params.camera.origin +
                gridPos.x * params.camera.u * SCALE
              + gridPos.y * params.camera.v * SCALE;
            
        }
        // tmin: Distance minimum / maximum d'intersection
        const float tmin = 0.0f, tmax = 1e16f;

        const float rayTime = 0.0f; // Non-utilisé
        const unsigned int rayFlags = OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT;


        // Masque de visibilité
        // Une intersection est trouvée si au moyen un bit en commun est utilisé
        // Comme on utilisé pas de masque, on met tous les bits à 1 pour collisionner contre toutes les géométries
        // OptixVisibilityMask est un int mais seulement 8 bits sont vraiment utilisés
        // Si on met à 0, il ne s'affiche rien car aucun triangle ne collisionne
        const OptixVisibilityMask visibilityMask(255);

        // Non-utilisés
        const unsigned int sbtOffset = 0, sbtStride = 0; // ?

        // Indice du programm "__miss__"
        // On n'en contient qu'un seul, donc l'indice 0.
        const unsigned int missSbtIndex = 0;


        unsigned int payload_0;

        optixTrace(
            params.traversableHandle,
            rayOrigin,
            rayDirection,
            tmin,
            tmax,
            rayTime,
            visibilityMask,
            rayFlags,
            sbtOffset,
            sbtStride,
            missSbtIndex,
            payload_0
        );
        
        uchar3& pixel = *params.at(idx.x, idx.y);
        
        uchar3 rgb = int_as_uchar3(payload_0);
        pixel = rgb;
    }

    __global__ void __anyhit__my_program()
    {
    }

    __global__ void __closesthit__my_program()
    {
        const Point *pointBase = *reinterpret_cast<const Point**>(optixGetSbtDataPointer());
        const size_t primitiveIndex = optixGetPrimitiveIndex();
        const uchar3 rgb = pointBase[primitiveIndex].col;

        optixSetPayload_0(uchar3_as_int(rgb));
    }

    __global__ void __miss__my_program()
    {
        // Exécuté quand le rayon ne trouve pas de collision
        // Envoyer la couleur de fond
        optixSetPayload_0(uchar3_as_int(make_uchar3(0, 0, 0)));
    }

    #define USE_SPHERE 1

    #if USE_SPHERE

    __host__ __device__ inline int
    resolve_2nd_equation(float solutions[2], float a, float b, float c)
    {
        int solutionsCount = 0;
        const float delta = b*b - 4*a*c;

        if(delta < 0.0f)
        {
            // Pas de solution réelle
        }
        else if(delta == 0.0f)
        {
            // 1 solution
            solutionsCount = 1;
            solutions[0] = -b / (2.0f * a);
        }
        else
        {
            // 2 solutions
            solutionsCount = 2;
            solutions[0] = (-b - sqrt(delta)) / (2.0f * a);
            solutions[1] = (-b + sqrt(delta)) / (2.0f * a);
        }

        return solutionsCount;
    }

    /**
    * @brief Programme d'intersection avec des sphères provenant de l'exemple du SDK (cuda/sphere.cu)
    * changé avec les structures de données pour s'adapter pour ce code.
    * 
    */
    __global__ void __intersection__my_program()
    {
        const Point *pointBase = *reinterpret_cast<const Point**>(optixGetSbtDataPointer());
        const size_t primitiveIndex = optixGetPrimitiveIndex();
        const Point& point = pointBase[primitiveIndex];

        const float3 ray_orig = optixGetWorldRayOrigin();
        const float3 ray_dir  = optixGetWorldRayDirection();
        const float  ray_tmin = optixGetRayTmin();
        const float  ray_tmax = optixGetRayTmax();

        const float3 c = point.pos;
        const float3 A = ray_orig;
        const float3 n = ray_dir;
        const float  r = point.r;

        const float eq_a = lengthSquared(n);
        const float eq_b = -2.0f * dot(c - A, n);
        const float eq_c = lengthSquared(c - A) - r*r;

        float solutions[2];

        switch(resolve_2nd_equation(solutions, eq_a, eq_b, eq_c))
        {
            case 0:
                // Pas d'intersection
                break;
            
            case 1:
                {
                    // 1 intersection
                    const float t = solutions[0];
                    if(t > ray_tmin && t < ray_tmax) {
                        optixReportIntersection(t, 0);
                    }
                }
                break;

            case 2:
                {
                    // 2 intersections
                    // On reporte toujours la plus proche d'abord,
                    // Donc pour le t plus petit
                    float t_near, t_far;
                    if(solutions[0] < solutions[1])
                    {
                        t_near = solutions[0];
                        t_far = solutions[1];
                    }
                    else
                    {
                        t_near = solutions[1];
                        t_far = solutions[0];
                    }
                    
                    // Les conditions ne servent à rien,
                    // La doc indique explicitement que optixReportIntersection()
                    // ne fait rien si t n'est pas dans la range

                    //if(t_near > ray_tmin && t_near < ray_tmax)
                    {
                        optixReportIntersection(t_near, 0);
                    }
                    //if(t_far > ray_tmin && t_far < ray_tmax)
                    {
                        //optixReportIntersection(t_far, 0);
                    }
                }
                break;
        }


        /*

        const float3 O      = ray_orig - point.pos;
        const float  l      = 1.0f / length( ray_dir );
        const float3 D      = ray_dir * l;
        const float  radius = point.r;

        float b    = dot( O, D );
        float c    = dot( O, O ) - radius * radius;
        float disc = b * b - c;
        if( disc > 0.0f )
        {
            float sdisc        = sqrtf( disc );
            float root1        = ( -b - sdisc );
            float root11       = 0.0f;
            bool  check_second = true;

            const bool do_refine = fabsf( root1 ) > ( 10.0f * radius );

            if( do_refine )
            {
                // refine root1
                float3 O1 = O + root1 * D;
                b         = dot( O1, D );
                c         = dot( O1, O1 ) - radius * radius;
                disc      = b * b - c;

                if( disc > 0.0f )
                {
                    sdisc  = sqrtf( disc );
                    root11 = ( -b - sdisc );
                }
            }

            float  t;
            float3 normal;
            t = ( root1 + root11 ) * l;
            if( t > ray_tmin && t < ray_tmax )
            {
                normal = ( O + ( root1 + root11 ) * D ) / radius;
                if( optixReportIntersection( t, 0))
                    check_second = false;
            }

            if( check_second )
            {
                float root2 = ( -b + sdisc ) + ( do_refine ? root1 : 0 );
                t           = root2 * l;
                normal      = ( O + root2 * D ) / radius;
                if( t > ray_tmin && t < ray_tmax )
                    optixReportIntersection( t, 0);
            }
        }
        */
    }

    #else

    __global__ void __intersection__my_program()
    {
        // 0: le type de collision défini par l'utilisateur, non-utilisé
        
        const float t = optixGetRayTmin();
        optixReportIntersection(t, 0);
    }

    #endif
}